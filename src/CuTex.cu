#include "hip/hip_runtime.h"
#include "CuTex.cuh"
#include <cstring>   // std::memset
#include <cmath>     // std::abs
#include <iostream>
#include "cumath.cuh"

static inline void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error (" << msg << "): " << hipGetErrorString(err) << "\n";
    }
}

CuTex::CuTex()
{
    tex_obj = 0;
    dev_ptr = nullptr;
    width = height = 0;
}

// Constructor to create linear memory and a texture object from it.
CuTex::CuTex(int _width, int _height,
            hipChannelFormatDesc _cudaChannelFormatDesc,
             hipTextureAddressMode _cudaTextureAddressMode,
             hipTextureFilterMode _cudaTextureFilterMode)
{
    tex_obj = 0;
    dev_ptr = nullptr;

    width = _width;
    height = _height;

    if (width <= 0 || height <= 0) {
        std::cerr << "CuTex: width/height must be > 0\n";
        return;
    }

    // Allocate linear device memory for the float buffer.
    hipError_t err = hipMalloc(&dev_ptr, static_cast<size_t>(width) * height * sizeof(float));
    if (err != hipSuccess) {
        checkCudaError(err, "hipMalloc");
        dev_ptr = nullptr;
        return;
    }

    // Describe the linear resource for texture creation.
    hipResourceDesc resDesc;
    std::memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = dev_ptr;
    resDesc.res.linear.desc = _cudaChannelFormatDesc;
    resDesc.res.linear.sizeInBytes = static_cast<size_t>(width) * height * sizeof(float);

    // Create texture object from the linear memory.
    hipTextureDesc texDesc;
    std::memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = _cudaTextureAddressMode;
    texDesc.addressMode[1] = _cudaTextureAddressMode;
    texDesc.filterMode = _cudaTextureFilterMode;
    texDesc.readMode = hipReadModeElementType; // Read as element type (float)
    texDesc.normalizedCoords = 0; // Use unnormalized coordinates for direct mapping
    
    err = hipCreateTextureObject(&tex_obj, &resDesc, &texDesc, nullptr);
    if (err != hipSuccess) {
        checkCudaError(err, "hipCreateTextureObject");
        hipFree(dev_ptr);
        dev_ptr = nullptr;
        tex_obj = 0;
        return;
    }
}

CuTex::~CuTex()
{
    if (tex_obj != 0) {
        hipError_t e = hipDestroyTextureObject(tex_obj);
        if (e != hipSuccess) checkCudaError(e, "hipDestroyTextureObject");
        tex_obj = 0;
    }
    if (dev_ptr != nullptr) {
        hipError_t e = hipFree(dev_ptr);
        if (e != hipSuccess) checkCudaError(e, "hipFree");
        dev_ptr = nullptr;
    }
}

void CuTex::readback(float* buffer)
{
    if (!buffer) {
        std::cerr << "CuTex::readback - null host buffer\n";
        return;
    }
    if (!dev_ptr) {
        std::cerr << "CuTex::readback - no dev_ptr allocation\n";
        return;
    }
    size_t totalBytes = static_cast<size_t>(width) * height * sizeof(float);
    hipError_t err = hipMemcpy(
        buffer,
        dev_ptr,
        totalBytes,
        hipMemcpyDeviceToHost
    );
    if (err != hipSuccess) {
        checkCudaError(err, "hipMemcpy (readback)");
    }
}
