#include "hip/hip_runtime.h"
#include "CuTex.cuh"
#include <cstring>   // std::memset
#include <cmath>     // std::abs
#include <iostream>
#include "cumath.cuh"

static inline void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error (" << msg << "): " << hipGetErrorString(err) << "\n";
    }
}

CuTex::CuTex()
{
    tex_obj = 0;
    dev_ptr = nullptr;
    width = height = 0;
}

CuTex::CuTex(CuTex&& other) noexcept
{
    tex_obj = other.tex_obj;
    dev_ptr = other.dev_ptr;
    width = other.width;
    height = other.height;

    // leave other in a safe-to-destruct state
    other.tex_obj = 0;
    other.dev_ptr = nullptr;
    other.width = other.height = 0;
}

// Move assignment
CuTex& CuTex::operator=(CuTex&& other) noexcept
{
    if (this != &other) {
        // free our resources first
        if (tex_obj != 0) { hipDestroyTextureObject(tex_obj); tex_obj = 0; }
        if (dev_ptr != nullptr) { hipFree(dev_ptr); dev_ptr = nullptr; }

        // steal
        tex_obj = other.tex_obj;
        dev_ptr = other.dev_ptr;
        width = other.width;
        height = other.height;

        // null out other
        other.tex_obj = 0;
        other.dev_ptr = nullptr;
        other.width = other.height = 0;
    }
    return *this;
}

// Constructor to create linear memory and a texture object from it.
CuTex::CuTex(int _width, int _height,
             hipChannelFormatDesc _cudaChannelFormatDesc,
             hipTextureAddressMode _cudaTextureAddressMode,
             hipTextureFilterMode _cudaTextureFilterMode)
{
    tex_obj = 0;
    dev_ptr = nullptr;
    width = _width;
    height = _height;

    if (width <= 0 || height <= 0) {
        std::cerr << "CuTex: width/height must be > 0\n";
        return;
    }

    size_t pitch = 0;
    // Allocate pitched device memory
    hipError_t err = hipMallocPitch(reinterpret_cast<void**>(&dev_ptr),
                                      &pitch,
                                      width * sizeof(float),
                                      height);
    if (err != hipSuccess) {
        checkCudaError(err, "hipMallocPitch");
        dev_ptr = nullptr;
        return;
    }

    // Zero the memory
    err = hipMemset2D(dev_ptr, pitch, 0, width * sizeof(float), height);
    if (err != hipSuccess) {
        checkCudaError(err, "hipMemset2D");
        hipFree(dev_ptr);
        dev_ptr = nullptr;
        return;
    }

    // Describe the pitched 2D resource for texture creation.
    hipResourceDesc resDesc;
    std::memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = dev_ptr;
    resDesc.res.pitch2D.desc = _cudaChannelFormatDesc;
    resDesc.res.pitch2D.width = width;
    resDesc.res.pitch2D.height = height;
    resDesc.res.pitch2D.pitchInBytes = pitch;

    // Texture description
    hipTextureDesc texDesc;
    std::memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = _cudaTextureAddressMode;
    texDesc.addressMode[1] = _cudaTextureAddressMode;
    texDesc.filterMode = _cudaTextureFilterMode;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 1; // use unnormalized coords (x in [0,width), y in [0,height))

    // Create texture object
    err = hipCreateTextureObject(&tex_obj, &resDesc, &texDesc, nullptr);
    if (err != hipSuccess) {
        checkCudaError(err, "hipCreateTextureObject");
        hipFree(dev_ptr);
        dev_ptr = nullptr;
        tex_obj = 0;
        return;
    }
}


CuTex::~CuTex()
{
    if (tex_obj != 0) {
        hipError_t e = hipDestroyTextureObject(tex_obj);
        if (e != hipSuccess) checkCudaError(e, "hipDestroyTextureObject");
        tex_obj = 0;
    }
    if (dev_ptr != nullptr) {
        hipError_t e = hipFree(dev_ptr);
        if (e != hipSuccess) checkCudaError(e, "hipFree");
        dev_ptr = nullptr;
    }
}

void CuTex::readback(float* buffer)
{
    if (!buffer) {
        std::cerr << "CuTex::readback - null host buffer\n";
        return;
    }
    if (!dev_ptr) {
        std::cerr << "CuTex::readback - no dev_ptr allocation\n";
        return;
    }
    size_t totalBytes = static_cast<size_t>(width) * height * sizeof(float);
    hipError_t err = hipMemcpy(
        buffer,
        dev_ptr,
        totalBytes,
        hipMemcpyDeviceToHost
    );
    if (err != hipSuccess) {
        checkCudaError(err, "hipMemcpy (readback)");
    }
}
