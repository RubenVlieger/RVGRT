#include "Framebuffer.cuh"
#include <hip/hip_runtime.h>
#include <iostream>

Framebuffer::Framebuffer()
    : width(0), height(0), d_pixels(nullptr) {
    
}   

Framebuffer::~Framebuffer() {

}

void Framebuffer::Allocate(int w, int h) {
    if (d_pixels) {
        Free(); // free old buffer
    }

    width = w;
    height = h;

    size_t size = width * height * sizeof(uint32_t);
    hipError_t err = hipMalloc(&d_pixels, size);
    if (err != hipSuccess) {
        std::cerr << "CUDA malloc failed: " << hipGetErrorString(err) << std::endl;
        d_pixels = nullptr;
    } else {
        // Optionally clear framebuffer
        hipMemset(d_pixels, 0, size);
    }
}

void Framebuffer::Free() {
    if (d_pixels) {
        hipFree(d_pixels);
        d_pixels = nullptr;
    }
}

std::vector<uint32_t> Framebuffer::readback() const {
    std::vector<uint32_t> cpuBuffer(width * height);
    if (d_pixels) {
        hipMemcpy(cpuBuffer.data(), d_pixels, width * height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }
    return cpuBuffer;
}

void Framebuffer::readback(uint32_t* buffer) const {
    if (d_pixels) {
        hipMemcpy(buffer, d_pixels, width * height * sizeof(uint32_t), hipMemcpyDeviceToHost);
    }
}

